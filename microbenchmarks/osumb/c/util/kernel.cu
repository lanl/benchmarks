
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2002-2021 the Network-Based Computing Laboratory
 * (NBCL), The Ohio State University.
 *
 * Contact: Dr. D. K. Panda (panda@cse.ohio-state.edu)
 *
 * For detailed copyright and licensing information, please refer to the
 * copyright file COPYRIGHT in the top level OMB directory.
 */

__global__ void compute_kernel(float a, float *x, float *y, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int count = 0;

    if (i < N) {
        for (count = 0; count < (N / 8); count++) {
            y[i] = a * x[i] + y[i];
        }
    }
}

__global__ void touch_managed_kernel(char *buf, size_t len)
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
        buf[i] = buf[i] + 1;
    }
}

__global__ void empty_kernel(char *buf, size_t len) {}

extern "C" void call_kernel(float a, float *d_x, float *d_y, int N,
                            hipStream_t *stream)
{
    compute_kernel<<<(N + 255) / 256, 256, 0, *stream>>>(a, d_x, d_y, N);
}

extern "C" void call_touch_managed_kernel(char *buf, size_t length,
                                          hipStream_t *stream)
{
    touch_managed_kernel<<<(length + 255) / 256, 256, 0, *stream>>>(buf,
                                                                    length);
}

extern "C" void call_empty_kernel(char *buf, size_t length,
                                  hipStream_t *stream)
{
    empty_kernel<<<(length + 255) / 256, 256, 0, *stream>>>(buf, length);
}
